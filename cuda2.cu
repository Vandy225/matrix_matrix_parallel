#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

double r8_uniform_01 ( int *seed ){
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * ( *seed - k * 127773 ) - k * 2836;

  if ( *seed < 0 )
  {
    *seed = *seed + 2147483647;
  }

  r = ( double ) ( *seed ) * 4.656612875E-10;

  return r;
}

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
int width;
int height;
float* elements;
int stride;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);





// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

void MatMul(const Matrix A, const Matrix B, Matrix C) {

unsigned long long l = A.height;
unsigned long long m = A.height;
unsigned long long n = A.height;

// Load A and B to device memory
Matrix d_A;
d_A.width = A.width;
d_A.height = A.height;
size_t size = A.width * A.height * sizeof(float);
hipError_t err = hipMalloc(&d_A.elements, size);
printf("CUDA malloc A: %s\n",hipGetErrorString(err));
err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
printf("Copy A to device: %s\n",hipGetErrorString(err));


Matrix d_B;
d_B.width = B.width;
d_B.height = B.height;
size = B.width * B.height * sizeof(float);
err = hipMalloc(&d_B.elements, size);
printf("CUDA malloc B: %s\n",hipGetErrorString(err));
err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
printf("Copy B to device: %s\n",hipGetErrorString(err));


// Allocate C in device memory
Matrix d_C;
d_C.width = C.width;
d_C.height = C.height;
size = C.width * C.height * sizeof(float);
err = hipMalloc(&d_C.elements, size);
printf("CUDA malloc C: %s\n",hipGetErrorString(err));
// Invoke kernel
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x,
(A.height + dimBlock.y - 1) / dimBlock.y);

float time_elapsed;

//struct timeval t1, t2;

//gettimeofday(&t1, 0);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
err = hipDeviceSynchronize();

hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time_elapsed,start,stop);
hipEventDestroy(start);
hipEventDestroy(stop);

//gettimeofday(&t2, 0);

//time_elapsed=(1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec);

  unsigned long long ops = l * l * ( 2 * l );

time_elapsed = time_elapsed/1000; //change into seconds
  
  double rate = ( double ) ( ops ) / time_elapsed / 1000000.0;

  printf ( "\n" );
  printf ( "CUDA matrix multiplication unoptimized serial.\n" );
  //printf ( "Number of threads: %d\n", num_t );
  printf ( "  A(LxN) = B(LxM) * C(MxN).\n" );
  printf ( "  L = %llu\n", l );
  printf ( "  M = %llu\n", m );
  printf ( "  N = %llu\n", n );
  printf ( "  Floating point OPS roughly %llu\n", ops );
  printf ( "  Elapsed time dT = %f\n", time_elapsed);
  printf ( "  Rate = MegaOPS/dT = %f\n", rate );

printf("Run kernel: %s\n", hipGetErrorString(err));
// Read C from device memory
err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
printf("Copy C off of device: %s\n",hipGetErrorString(err));
// Free device memory
hipFree(d_A.elements);
hipFree(d_B.elements);
// hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
// Each thread computes one element of C
// by accumulating results into Cvalue
float Cvalue = 0.0;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if(row > A.height || col > B.width) return;

for (int e = 0; e < A.width; ++e)
Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
C.elements[row * C.width + col] = Cvalue;
}



// Matrix multiplication kernel called by MatMul()
__global__ void ShareMatMulKernel(Matrix A, Matrix B, Matrix C) {
// Block row and column
int blockRow = blockIdx.y;
int blockCol = blockIdx.x;
// Each thread block computes one sub-matrix Csub of C
Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
// Each thread computes one element of Csub
// by accumulating results into Cvalue
float Cvalue = 0.0;
// Thread row and column within Csub
int row = threadIdx.y;
int col = threadIdx.x;
// Loop over all the sub-matrices of A and B that are
// required to compute Csub
// Multiply each pair of sub-matrices together
// and accumulate the results
for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
// Get sub-matrix Asub of A
Matrix Asub = GetSubMatrix(A, blockRow, m);
// Get sub-matrix Bsub of B
Matrix Bsub = GetSubMatrix(B, m, blockCol);
// Shared memory used to store Asub and Bsub respectively
__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
// Load Asub and Bsub from device memory to shared memory
// Each thread loads one element of each sub-matrix
As[row][col] = GetElement(Asub, row, col);
Bs[row][col] = GetElement(Bsub, row, col);
// Synchronize to make sure the sub-matrices are loaded
// before starting the computation
__syncthreads();
// Multiply Asub and Bsub together
for (int e = 0; e < BLOCK_SIZE; ++e)
Cvalue += As[row][e] * Bs[e][col];
// Synchronize to make sure that the preceding
// computation is done before loading two new
// sub-matrices of A and B in the next iteration
__syncthreads();
}
// Write Csub to device memory
// Each thread writes one element
SetElement(Csub, row, col, Cvalue);
}



// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void ShareMatMul(const Matrix A, const Matrix B, Matrix C) {
// Load A and B to device memory
Matrix d_A;
d_A.width = d_A.stride = A.width;
d_A.height = A.height;
size_t size = A.width * A.height * sizeof(float);
hipError_t err = hipMalloc(&d_A.elements, size);
printf("CUDA malloc A: %s\n",hipGetErrorString(err));
hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
Matrix d_B;
d_B.width = d_B.stride = B.width;
d_B.height = B.height;
size = B.width * B.height * sizeof(float);
err = hipMalloc(&d_B.elements, size);
printf("CUDA malloc B: %s\n",hipGetErrorString(err));

hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
Matrix d_C;
d_C.width = d_C.stride = C.width;
d_C.height = C.height;
size = C.width * C.height * sizeof(float);
err = hipMalloc(&d_C.elements, size);
printf("CUDA malloc C: %s\n",hipGetErrorString(err));
// Invoke kernel
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
ShareMatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
err = hipDeviceSynchronize();
printf("Run kernel: %s\n", hipGetErrorString(err));
// Read C from device memory
err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
printf("Copy C off of device: %s\n",hipGetErrorString(err));
// Free device memory
hipFree(d_A.elements);
hipFree(d_B.elements);
hipFree(d_C.elements);
}


// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
return A.elements[row * A.stride + col];
}


// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
A.elements[row * A.stride + col] = value;
}



// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is

// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
Matrix Asub;
Asub.width = BLOCK_SIZE;
Asub.height = BLOCK_SIZE;
Asub.stride = A.stride;
Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
return Asub;
}














// Usage: multNoShare a1 a2 b2
int main(int argc, char* argv[]){
Matrix A, B, C;
int a1, a2, b1, b2;
// Read some values from the commandline
a1 = atoi(argv[1]); /* Height of A */
a2 = atoi(argv[2]); /* Width of A */
b1 = a2; /* Height of B */
b2 = atoi(argv[3]); /* Width of B */
A.height = a1;
A.width = a2;
A.elements = (float*)malloc(A.width * A.height * sizeof(float));
B.height = b1;
B.width = b2;
B.elements = (float*)malloc(B.width * B.height * sizeof(float));
C.height = A.height;
C.width = B.width;
C.elements = (float*)malloc(C.width * C.height * sizeof(float));

int seed=123456789;


for(int i = 0; i < A.height; i++)
for(int j = 0; j < A.width; j++)
//A.elements[i*A.width + j] = (float)(arc4random() % 3);
A.elements[i*A.width + j] = (float) (r8_uniform_01 ( &seed ));
for(int i = 0; i < B.height; i++)
for(int j = 0; j < B.width; j++)
//B.elements[i*B.width + j] = (float)(arc4random() % 2);
B.elements[i*B.width + j] = (float) (r8_uniform_01 ( &seed ));
MatMul(A, B, C);
ShareMatMul(A,B,C);

}
