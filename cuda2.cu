#include "hip/hip_runtime.h"
#include <stdio.h>

double r8_uniform_01 ( int *seed ){
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * ( *seed - k * 127773 ) - k * 2836;

  if ( *seed < 0 )
  {
    *seed = *seed + 2147483647;
  }

  r = ( double ) ( *seed ) * 4.656612875E-10;

  return r;
}

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
int width;
int height;
float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);





// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

void MatMul(const Matrix A, const Matrix B, Matrix C) {

int l = A.height;
int m = A.height;
int n = A.height;

// Load A and B to device memory
Matrix d_A;
d_A.width = A.width;
d_A.height = A.height;
size_t size = A.width * A.height * sizeof(float);
hipError_t err = hipMalloc(&d_A.elements, size);
printf("CUDA malloc A: %s\n",hipGetErrorString(err));
err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
printf("Copy A to device: %s\n",hipGetErrorString(err));


Matrix d_B;
d_B.width = B.width;
d_B.height = B.height;
size = B.width * B.height * sizeof(float);
err = hipMalloc(&d_B.elements, size);
printf("CUDA malloc B: %s\n",hipGetErrorString(err));
err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
printf("Copy B to device: %s\n",hipGetErrorString(err));


// Allocate C in device memory
Matrix d_C;
d_C.width = C.width;
d_C.height = C.height;
size = C.width * C.height * sizeof(float);
err = hipMalloc(&d_C.elements, size);
printf("CUDA malloc C: %s\n",hipGetErrorString(err));
// Invoke kernel
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x,
(A.height + dimBlock.y - 1) / dimBlock.y);

float time_elapsed;
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
  
hipEventRecord(start,0);
MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
err = hipDeviceSynchronize();
hipEventRecord(stop,0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time_elapsed,start,stop);

  unsigned long long ops = l * l * ( 2 * l );
  
  double rate = ( double ) ( ops ) / time_elapsed*1000 / 1000000.0;

  printf ( "\n" );
  printf ( "CUDA matrix multiplication unoptimized serial.\n" );
  #printf ( "Number of threads: %d\n", num_t );
  printf ( "  A(LxN) = B(LxM) * C(MxN).\n" );
  printf ( "  L = %llu\n", l );
  printf ( "  M = %llu\n", m );
  printf ( "  N = %llu\n", n );
  printf ( "  Floating point OPS roughly %llu\n", ops );
  printf ( "  Elapsed time dT = %f\n", time_elapsed*1000 );
  printf ( "  Rate = MegaOPS/dT = %f\n", rate );

printf("Run kernel: %s\n", hipGetErrorString(err));
// Read C from device memory
err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
printf("Copy C off of device: %s\n",hipGetErrorString(err));
// Free device memory
hipFree(d_A.elements);
hipFree(d_B.elements);
// hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
// Each thread computes one element of C
// by accumulating results into Cvalue
float Cvalue = 0.0;
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if(row > A.height || col > B.width) return;

for (int e = 0; e < A.width; ++e)
Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
C.elements[row * C.width + col] = Cvalue;
}
// Usage: multNoShare a1 a2 b2
int main(int argc, char* argv[]){
Matrix A, B, C;
int a1, a2, b1, b2;
// Read some values from the commandline
a1 = atoi(argv[1]); /* Height of A */
a2 = atoi(argv[2]); /* Width of A */
b1 = a2; /* Height of B */
b2 = atoi(argv[3]); /* Width of B */
A.height = a1;
A.width = a2;
A.elements = (float*)malloc(A.width * A.height * sizeof(float));
B.height = b1;
B.width = b2;
B.elements = (float*)malloc(B.width * B.height * sizeof(float));
C.height = A.height;
C.width = B.width;
C.elements = (float*)malloc(C.width * C.height * sizeof(float));

int seed=123456789;


for(int i = 0; i < A.height; i++)
for(int j = 0; j < A.width; j++)
#A.elements[i*A.width + j] = (float)(arc4random() % 3);
A.elements[i*A.width + j] = (float) (r8_uniform_01 ( &seed ));
for(int i = 0; i < B.height; i++)
for(int j = 0; j < B.width; j++)
#B.elements[i*B.width + j] = (float)(arc4random() % 2);
B.elements[i*B.width + j] = (float) (r8_uniform_01 ( &seed ));
MatMul(A, B, C);

}
